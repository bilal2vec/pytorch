#include "stdio.h"

#include <vector>
#include <set>

#include <hip/hip_runtime.h>


#define N 1024 * 1024

__global__ void init(float* dst, float* src) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] = src[idx];
}

__global__ void add(float* dst, float* src) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] += src[idx];
}

int main(void) {
  float *h_src_1, *h_src_2, *h_dst, *d_src_1, *d_src_2, *d_dst;
  std::vector<hipGraphNode_t> _node_list;

  hipStream_t stream;
  hipGraph_t graph;
  hipGraphExec_t instance;

  hipStreamCreate(&stream);

  h_src_1 = (float*)malloc(sizeof(float) * N);
  h_src_2 = (float*)malloc(sizeof(float) * N);
  h_dst = (float*)malloc(sizeof(float) * N);

  hipMalloc((void**)&d_src_1, sizeof(float) * N);
  hipMalloc((void**)&d_src_2, sizeof(float) * N);
  hipMalloc((void**)&d_dst, sizeof(float) * N);

  for (size_t i = 0; i < N; i++) {
    h_src_1[i] = 1;
    h_src_2[i] = 2;
    h_dst[i] = 0.0;
  }

  hipMemcpyAsync(d_src_1, h_src_1, sizeof(float) * N, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_src_2, h_src_2, sizeof(float) * N, hipMemcpyHostToDevice, stream);

  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  
  init<<<1024, 1024, 0, stream>>>(d_dst, d_src_1);
  add<<<1024, 1024, 0, stream>>>(d_dst, d_dst);

  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiateWithFlags(&instance, graph);

  hipGraphLaunch(instance, stream);
  
  hipMemcpyAsync(h_dst, d_dst, sizeof(float) * N, hipMemcpyDeviceToHost);
  hipStreamSynchronize(stream);

  printf("%f\n", h_dst[0]);
  printf("%f\n", h_dst[1024]);

  size_t n_nodes;
  std::set<float*> dynamic_input_ptrs;

  dynamic_input_ptrs.insert(d_src_1); 


  hipGraphGetNodes(graph, nullptr, &n_nodes);

  std::vector<hipGraphNode_t> nodes(n_nodes);
  hipGraphGetNodes(graph, &nodes[0], &n_nodes);

  for (auto node : nodes) {
    hipKernelNodeParams p;
    hipGraphKernelNodeGetParams(node, &p);
    float** dst = ((float***)p.kernelParams)[0];
    float** src = ((float***)p.kernelParams)[1];

    printf("%p %p\n", *dst, *src);
  }

  printf("d_dst: %p d_src_1: %p\n", d_dst, d_src_1);

  return 0;
}
