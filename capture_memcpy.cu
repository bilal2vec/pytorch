#include "stdio.h"
#include <vector>

#include <hip/hip_runtime.h>


#define N 1024 * 1024

__global__ void init(float* dst, float* src) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] = src[idx];
}

__global__ void add(float* dst, float* src) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] += src[idx];
}

int main(void) {
  float *h_src, *h_dst, *d_src, *d_dst;
  std::vector<hipGraphNode_t> _node_list;

  hipStream_t stream;
  hipGraph_t graph;
  hipGraphExec_t instance;

  hipStreamCreate(&stream);

  h_src = (float*)malloc(sizeof(float) * N);
  h_dst = (float*)malloc(sizeof(float) * N);

  hipMalloc((void**)&d_src, sizeof(float) * N);
  hipMalloc((void**)&d_dst, sizeof(float) * N);

  for (size_t i = 0; i < N; i++) {
    h_src[i] = 1;
    h_dst[i] = 0.0;
  }

  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  
  hipMemcpyAsync(d_src, h_src, sizeof(float) * N, hipMemcpyHostToDevice, stream);
  
  init<<<1024, 1024, 0, stream>>>(d_dst, d_src);
  add<<<1024, 1024, 0, stream>>>(d_dst, d_dst);
  add<<<1024, 1024, 0, stream>>>(d_dst, d_dst);
  add<<<1024, 1024, 0, stream>>>(d_dst, d_dst);
  
  hipStreamEndCapture(stream, &graph);
  hipStreamSynchronize(stream);
  hipGraphInstantiateWithFlags(&instance, graph);

  hipGraphLaunch(instance, stream);
  hipMemcpyAsync(h_dst, d_dst, sizeof(float) * N, hipMemcpyDeviceToHost);
  hipStreamSynchronize(stream);

  printf("%f\n", h_dst[0]);
  printf("%f\n", h_dst[1024]);

  return 0;
}
