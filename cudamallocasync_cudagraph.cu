#include <iostream>

#include <hip/hip_runtime.h>


#define N 8 

__global__ void init(float* dst) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    dst[i] = 0.0f;
}

__global__ void add(float* dst) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    dst[i] = dst[i] + 1.0f;
}

int main(void) {
    hipStream_t stream;
    hipGraph_t graph;
    hipGraphExec_t instance;

    hipStreamCreate(&stream);

    float *src_h, *src_d;

    src_h = (float*)malloc(sizeof(float) * N); 
    for (size_t i = 0; i < N; i++) {
        src_h[i] = 0.0f;
    }

    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    //cudaMalloc((void**)&src_d, sizeof(float) * N);
    hipMallocAsync((void**)&src_d, sizeof(float) * N, stream);
    init<<<1, N, 0, stream>>>(src_d);
    add<<<1, N, 0, stream>>>(src_d);

    hipStreamEndCapture(stream, &graph);
    hipStreamSynchronize(stream);
    hipGraphInstantiateWithFlags(&instance, graph);

    //cudaGraphLaunch(instance, stream);
    hipMemcpyAsync(src_h, src_d, sizeof(float) * N, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    for (size_t i = 0; i < N; i++) {
        std::cout << src_h[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}