#include "hip/hip_runtime.h"
#include "stdio.h"
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 1024 * 1024

__global__ void init(float* dst, float* src) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] = src[idx];
}

__global__ void add(float* dst, float* src) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] += src[idx];
}

int main(void) {
  float *h_src_1, *h_src_2, *h_dst, *d_src_1, *d_src_2, *d_dst;
  std::vector<hipGraphNode_t> _node_list;

  hipStream_t stream;
  hipGraph_t graph;
  hipGraphExec_t instance;

  hipStreamCreate(&stream);

  h_src_1 = (float*)malloc(sizeof(float) * N);
  h_src_2 = (float*)malloc(sizeof(float) * N);
  h_dst = (float*)malloc(sizeof(float) * N);

  hipMalloc((void**)&d_src_1, sizeof(float) * N);
  hipMalloc((void**)&d_src_2, sizeof(float) * N);
  hipMalloc((void**)&d_dst, sizeof(float) * N);

  for (size_t i = 0; i < N; i++) {
    h_src_1[i] = 1;
    h_src_2[i] = 2;
    h_dst[i] = 0.0;
  }

  hipMemcpyAsync(d_src_1, h_src_1, sizeof(float) * N, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_src_2, h_src_2, sizeof(float) * N, hipMemcpyHostToDevice, stream);

  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
 
  hipGraph_t graph_so_far;
  hipStreamCaptureStatus capture_status; 
  hipKernelNodeParams params;
  hipGraphNode_t init_kernel_node;
  const hipGraphNode_t* deps;
  size_t dep_count;

  params.func = reinterpret_cast<void*>(init);
  params.blockDim = {static_cast<unsigned int>(1024), 1, 1};
  params.gridDim = {static_cast<unsigned int>(1024), 1, 1};
  params.sharedMemBytes = 0;
  void* args_1[] = {&d_dst, &d_src_1};
  params.kernelParams = args_1;
  params.extra = nullptr;

  cudaStreamGetCaptureInfo_v2(stream, &capture_status, nullptr, &graph_so_far, &deps, &dep_count);

  hipGraphAddKernelNode(&init_kernel_node, graph_so_far, deps, dep_count, &params);
  _node_list.push_back(init_kernel_node);
  hipStreamUpdateCaptureDependencies(stream, &init_kernel_node, 1, 1);

//  add<<<1024, 1024, 0, stream>>>(d_dst, d_src_1);
  add<<<1024, 1024, 0, stream>>>(d_dst, d_dst);

  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiate(&instance, graph);

  hipGraphLaunch(instance, stream);
  hipMemcpyAsync(h_dst, d_dst, sizeof(float) * N, hipMemcpyDeviceToHost);
  hipStreamSynchronize(stream);

  printf("%f\n", h_dst[0]);
  printf("%f\n", h_dst[1024]);

  params.kernelParams[1] = &d_src_2;

  hipGraphExecKernelNodeSetParams(instance, _node_list[0], &params);

  hipGraphLaunch(instance, stream);
  hipMemcpyAsync(h_dst, d_dst, sizeof(float) * N, hipMemcpyDeviceToHost);
  hipStreamSynchronize(stream);

  printf("%f\n", h_dst[0]);
  printf("%f\n", h_dst[1024]);

  return 0;
}
