#include "stdio.h"
#include <vector>

#include <hip/hip_runtime.h>


#define N 1024 * 1024

__global__ void init(float* dst, float* src) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] = src[idx];
}

__global__ void add(float* dst, float* src) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  dst[idx] += src[idx];
}

int main(void) {
  float *h_src, *h_dst, *d_src_1, *d_src_2, *d_dst_1, *d_dst_2;
 
  hipStream_t stream;
  hipStreamCreate(&stream);

  h_src = (float*)malloc(sizeof(float) * N);
  h_dst = (float*)malloc(sizeof(float) * N);

  hipMalloc((void**)&d_src_1, sizeof(float) * N);
  hipMalloc((void**)&d_src_2, sizeof(float) * N);
  hipMalloc((void**)&d_dst_1, sizeof(float) * N);
  hipMalloc((void**)&d_dst_2, sizeof(float) * N);

  for (size_t i = 0; i < N; i++) {
    h_src[i] = 1;
    h_dst[i] = 0.0;
  }
 
  hipMemcpyAsync(d_src_1, h_src, sizeof(float) * N, hipMemcpyHostToDevice, stream);
  hipStreamSynchronize(stream);
  hipMemcpyAsync(d_src_2, d_src_1, sizeof(float) * N, hipMemcpyDeviceToDevice, stream);
  hipStreamSynchronize(stream);
  
  init<<<1024, 1024, 0, stream>>>(d_dst_1, d_src_2);
  add<<<1024, 1024, 0, stream>>>(d_dst_1, d_dst_1);
  add<<<1024, 1024, 0, stream>>>(d_dst_1, d_dst_1);
  add<<<1024, 1024, 0, stream>>>(d_dst_1, d_dst_1);

  hipStreamSynchronize(stream);
  hipMemcpyAsync(d_dst_2, d_dst_1, sizeof(float) * N, hipMemcpyDeviceToDevice, stream);
  hipStreamSynchronize(stream);
  hipMemcpyAsync(h_dst, d_dst_2, sizeof(float) * N, hipMemcpyDeviceToHost, stream);
  hipStreamSynchronize(stream);
 
  printf("%f\n", h_dst[0]);
  printf("%f\n", h_dst[1024]);

  return 0;

}
